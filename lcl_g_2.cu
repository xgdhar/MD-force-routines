#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<time.h>

__global__
void updateKernel(float kappa, float G, float rc, float pi, int Ncell, int* bin_count, float* bin_atom_ln, int bin_atom_len, float* nl_list_ln, int cell_len)
{

  int ic = blockIdx.x + blockIdx.y * gridDim.x;
  int thId = threadIdx.x + blockDim.x * blockIdx.x + blockIdx.y * gridDim.x * blockDim.x;
  int ip = thId - ic*27;
  //num_threads - num_of_atoms_in_cutoff_sphere = 10;

  int jn;
  float Zc_ip, x_ip, y_ip, z_ip; 
  float Zc_jp, x_jp, y_jp, z_jp;
  int c_ipart, c_jpart; 
  float n_x_sh, n_y_sh, n_z_sh;
  float dx, dy, dz, r, f1, f2, f3, fr;
  float G_r_k_p, G_r_k_m;
 
  float sqt_pi = 1.7724538509055159;

  if(ic < Ncell){
    if(ip < (ic*37+bin_count[ic])){

  //for(int ic = 0; ic < Ncell; ic++){
    // for(int ip = 0; ip < bin_count[ic]; ip++){
       
        //c_ipart = bin_atom_ln[ic*bin_atom_len+ip*8];
        //Zc_ip = bin_atom_ln[ic*bin_atom_len+ip*8+1];
        //x_ip = bin_atom_ln[ic*bin_atom_len+ip*8+2];
        //y_ip = bin_atom_ln[ic*bin_atom_len+ip*8+3];
        //z_ip = bin_atom_ln[ic*bin_atom_len+ip*8+4];

        c_ipart = bin_atom_ln[ip*8];
        Zc_ip = bin_atom_ln[ip*8+1];
        x_ip = bin_atom_ln[ip*8+2];
        y_ip = bin_atom_ln[ip*8+3];
        z_ip = bin_atom_ln[ip*8+4];

        for(int jc = 0; jc < 27; jc++){
           jn = (int) nl_list_ln[ic*cell_len+jc*4];
           for(int jp = 0; jp < bin_count[jn]; jp++){

              c_jpart = bin_atom_ln[jn*bin_atom_len+jp*8];
              Zc_jp = bin_atom_ln[jn*bin_atom_len+jp*8+1];
              x_jp = bin_atom_ln[jn*bin_atom_len+jp*8+2];
              y_jp = bin_atom_ln[jn*bin_atom_len+jp*8+3];
              z_jp = bin_atom_ln[jn*bin_atom_len+jp*8+4];
              
              n_x_sh = nl_list_ln[ic*cell_len+jc*4+1];         
              n_y_sh = nl_list_ln[ic*cell_len+jc*4+2];         
              n_z_sh = nl_list_ln[ic*cell_len+jc*4+3];         
              //if(ic == 0) {printf("c = %d, c_i = %d, c_j = %d\n", ic, c_ipart, c_jpart);}
              if(c_ipart != c_jpart){
              
                //if((ic == 0) && (ip == 0)) {printf("c = %d, c_i = %d, c_j = %d\n", ic, c_ipart, c_jpart);}
                dx = x_ip - (x_jp + n_x_sh);
                dy = y_ip - (y_jp + n_y_sh);
                dz = z_ip - (z_jp + n_z_sh);
                r = sqrt(dx*dx + dy*dy + dz*dz);

                if(r < rc){

                  G_r_k_p = G*r + 0.5*kappa/G;
                  G_r_k_m = G*r - 0.5*kappa/G; 
                  f1 = (0.5/(r*r)) * exp(kappa*r) * erfc(G*r + 0.5*kappa/G) * (1.0 - kappa*r);
                  f2 = (0.5/(r*r)) * exp(-kappa*r) * erfc(G*r - 0.5*kappa/G) * (1.0 + kappa*r);
                  f3 = (G/(sqt_pi*r)) * (exp(-G_r_k_p*G_r_k_p) * exp(kappa*r) + exp(-G_r_k_m*G_r_k_m) * exp(-kappa*r) );
                  fr = Zc_ip*Zc_jp*(f1 + f2 + f3);           
               
                  //bin_atom_ln[ic*bin_atom_len+ip*8+5] = bin_atom_ln[ic*bin_atom_len+ip*8+5] + (fr*dx/r);   
                  //bin_atom_ln[ic*bin_atom_len+ip*8+6] = bin_atom_ln[ic*bin_atom_len+ip*8+6] + (fr*dy/r);   
                  //bin_atom_ln[ic*bin_atom_len+ip*8+7] = bin_atom_ln[ic*bin_atom_len+ip*8+7] + (fr*dz/r);   
                  
                  bin_atom_ln[ip*8+5] = bin_atom_ln[ip*8+5] + (fr*dx/r);   
                  bin_atom_ln[ip*8+6] = bin_atom_ln[ip*8+6] + (fr*dy/r);   
                  bin_atom_ln[ip*8+7] = bin_atom_ln[ip*8+7] + (fr*dz/r);   
                  
                  //bin_atom_ln[jn*bin_atom_len+jp*8+5] = bin_atom_ln[jn*bin_atom_len+jp*8+5] - (fr*dx/r);   
                  //bin_atom_ln[jn*bin_atom_len+jp*8+6] = bin_atom_ln[jn*bin_atom_len+jp*8+6] - (fr*dy/r);   
                  //bin_atom_ln[jn*bin_atom_len+jp*8+7] = bin_atom_ln[jn*bin_atom_len+jp*8+7] - (fr*dz/r);   

                }

              }    


           }


        }

    }

  }



}


void update(float kappa, float G, float rc, float pi, int Ncell, int* bin_count, float* bin_atom_ln, int bin_atom_len, float* nl_list_ln, int cell_len)
{

 // int cN, cp, cxN, cyN, czN, cxsh, cysh, czsh;
  //float rsh_x, rsh_y, rsh_z;

  int jn;
  float Zc_ip, x_ip, y_ip, z_ip; 
  float Zc_jp, x_jp, y_jp, z_jp;
  int c_ipart, c_jpart; 
  float n_x_sh, n_y_sh, n_z_sh;
  float dx, dy, dz, r, f1, f2, f3, fr;


  for(int ic = 0; ic < Ncell; ic++){
     for(int ip = 0; ip < bin_count[ic]; ip++){
       
        c_ipart = bin_atom_ln[ic*bin_atom_len+ip*8];
        Zc_ip = bin_atom_ln[ic*bin_atom_len+ip*8+1];
        x_ip = bin_atom_ln[ic*bin_atom_len+ip*8+2];
        y_ip = bin_atom_ln[ic*bin_atom_len+ip*8+3];
        z_ip = bin_atom_ln[ic*bin_atom_len+ip*8+4];

        for(int jc = 0; jc < 27; jc++){
           jn = (int) nl_list_ln[ic*cell_len+jc*4];
           for(int jp = 0; jp < bin_count[jn]; jp++){

              c_jpart = bin_atom_ln[jn*bin_atom_len+jp*8];
              Zc_jp = bin_atom_ln[jn*bin_atom_len+jp*8+1];
              x_jp = bin_atom_ln[jn*bin_atom_len+jp*8+2];
              y_jp = bin_atom_ln[jn*bin_atom_len+jp*8+3];
              z_jp = bin_atom_ln[jn*bin_atom_len+jp*8+4];
              
              n_x_sh = nl_list_ln[ic*cell_len+jc*4+1];         
              n_y_sh = nl_list_ln[ic*cell_len+jc*4+2];         
              n_z_sh = nl_list_ln[ic*cell_len+jc*4+3];         
              //if(ic == 0) {printf("c = %d, c_i = %d, c_j = %d\n", ic, c_ipart, c_jpart);}
              if(c_ipart < c_jpart){
              
                //if(ic == 0) {printf("c = %d, c_i = %d, c_j = %d\n", ic, c_ipart, c_jpart);}
                dx = x_ip - (x_jp + n_x_sh);
                dy = y_ip - (y_jp + n_y_sh);
                dz = z_ip - (z_jp + n_z_sh);
                r = sqrt(dx*dx + dy*dy + dz*dz);

                if(r < rc){
                  f1 = (0.5/(r*r)) * exp(kappa*r) * erfc(G*r + 0.5*kappa/G) * (1.0 - kappa*r);
                  f2 = (0.5/(r*r)) * exp(-kappa*r) * erfc(G*r - 0.5*kappa/G) * (1.0 + kappa*r);
                  f3 = (G/(sqrt(pi)*r)) * (exp(-pow((G*r + 0.5*kappa/G),2)) * exp(kappa*r) + exp(-pow((G*r - 0.5*kappa/G),2)) * exp(-kappa*r) );
                  fr = Zc_ip*Zc_jp*(f1 + f2 + f3);           
               
                  bin_atom_ln[ic*bin_atom_len+ip*8+5] = bin_atom_ln[ic*bin_atom_len+ip*8+5] + (fr*dx/r);   
                  bin_atom_ln[ic*bin_atom_len+ip*8+6] = bin_atom_ln[ic*bin_atom_len+ip*8+6] + (fr*dy/r);   
                  bin_atom_ln[ic*bin_atom_len+ip*8+7] = bin_atom_ln[ic*bin_atom_len+ip*8+7] + (fr*dz/r);   
                  
                  //bin_atom_ln[ip*8+5] = bin_atom_ln[ip*8+5] + (fr*dx/r);   
                  //bin_atom_ln[ip*8+6] = bin_atom_ln[ip*8+6] + (fr*dy/r);   
                  //bin_atom_ln[ip*8+7] = bin_atom_ln[ip*8+7] + (fr*dz/r);   
                  bin_atom_ln[jn*bin_atom_len+jp*8+5] = bin_atom_ln[jn*bin_atom_len+jp*8+5] - (fr*dx/r);   
                  bin_atom_ln[jn*bin_atom_len+jp*8+6] = bin_atom_ln[jn*bin_atom_len+jp*8+6] - (fr*dy/r);   
                  bin_atom_ln[jn*bin_atom_len+jp*8+7] = bin_atom_ln[jn*bin_atom_len+jp*8+7] - (fr*dz/r);   

                }

              }    


           }


        }

     }

  }

}

int main()
{

  float kappa, G;
  float L, rc;
  float rcx, rcy, rcz;
 
  float diff;
  struct timespec start, end; 
  float diff_c;
  struct timespec start_c, end_c; 

  int ipart;
  int i, j;
  int c, cx, cy, cz;


  int N = 10000000;

  float const pi = 3.141592653589793;
  float const emp = -50;

  float **pos = (float **)malloc(N*sizeof(float *));
  for(i = 0; i < N; i++){
     pos[i] = (float *)malloc(3*sizeof(float));
  }

  float *Z = (float *)malloc(N*sizeof(float));
  for(i = 0; i < N; i++){
     Z[i] = 1.0;
  }
  
  FILE *file;
  file = fopen("pos_1e7.txt", "r");

  for(i = 0; i < N; i++){
     for(j = 0; j < 3; j++){

        if(!fscanf(file, "%f", &pos[i][j]))
          break;
     }
  }

  fclose(file);

  kappa = 0.1;
  G = 0.5;
  rc = 3.0;

  L = pow(4.0 * pi * N/3.0, 1.0/3.0);
  printf("L = %f\n", L);

  float Lx = L;
  float Ly = L;
  float Lz = L;

  int Lxd = (int) floor(Lx/rc);
  int Lyd = (int) floor(Ly/rc);
  int Lzd = (int) floor(Lz/rc);

  int Ncell = Lxd*Lyd*Lzd;

  printf("%d %d %d %d\n", Lxd, Lyd, Lzd, Ncell);

  rcx = Lx/Lxd;
  rcy = Ly/Lyd;
  rcz = Lz/Lzd;

  printf("%f %f %f\n",rcx, rcy, rcz);

  int bin_atom_c = (int) (1* pow(rc,3) + 10.0);
  printf("%d\n",bin_atom_c);
  const int atm_len = 8;
  int bin_atom_len = atm_len * bin_atom_c;
  float bin_atom[Ncell][atm_len*bin_atom_c];
  //float bin_atom_ln[Ncell*bin_atom_len];
  float *bin_atom_ln = (float *)malloc(Ncell*bin_atom_len*sizeof(float));

  for(i = 0; i < Ncell; i++){
     for(j = 0; j < bin_atom_c; j++){
        bin_atom[i][j] = emp;
        bin_atom_ln[i*bin_atom_len + j] = emp;
     }
  }

  int *bin_count = (int *)malloc(Ncell*sizeof(int));
  int bcount, bin_idx;
  for(i = 0; i < Ncell; i++){
     bin_count[i] = 0;
  }

  printf("bin_count[1] = %d\n", bin_count[1]);

  for(ipart = 0; ipart < N; ipart++){
     
     cx = (int) floor(pos[ipart][0]/rcx);
     cy = (int) floor(pos[ipart][1]/rcy);
     cz = (int) floor(pos[ipart][2]/rcz);

     c = cx + cy*Lxd + cz*Lxd*Lyd;

     //if(c == 1){printf("c = %d, ipart = %d, bin_count[%d] = %d\n", c, ipart, c, bin_count[c]);}

     bcount = bin_count[c];
     bin_idx = atm_len*bcount;
     bin_atom[c][bin_idx] = ipart; 
     //if(c == 1){printf("c = %d, ipart = %d, bin_count[%d] = %d, bin_idx = %d, bin_atom[%d][bin_idx] = %f\n", c, ipart, c, bin_count[c], bin_idx, c, bin_atom[c][bin_idx]);}
     bin_atom[c][bin_idx+1] = Z[ipart]; 
     bin_atom[c][bin_idx+2] = pos[ipart][0]; 
     bin_atom[c][bin_idx+3] = pos[ipart][1]; 
     bin_atom[c][bin_idx+4] = pos[ipart][2]; 
     bin_atom[c][bin_idx+5] = 0.0; 
     bin_atom[c][bin_idx+6] = 0.0; 
     bin_atom[c][bin_idx+7] = 0.0;
    
     bin_atom_ln[c*bin_atom_len+bin_idx] = ipart; 
     //if(c == 1){printf("c = %d, ipart = %d, bin_count[%d] = %d, bin_idx = %d, bin_atom[%d][bin_idx] = %f\n", c, ipart, c, bin_count[c], bin_idx, c, bin_atom[c][bin_idx]);}
     bin_atom_ln[c*bin_atom_len+bin_idx+1] = Z[ipart]; 
     bin_atom_ln[c*bin_atom_len+bin_idx+2] = pos[ipart][0]; 
     bin_atom_ln[c*bin_atom_len+bin_idx+3] = pos[ipart][1]; 
     bin_atom_ln[c*bin_atom_len+bin_idx+4] = pos[ipart][2]; 
     bin_atom_ln[c*bin_atom_len+bin_idx+5] = 0.0; 
     bin_atom_ln[c*bin_atom_len+bin_idx+6] = 0.0; 
     bin_atom_ln[c*bin_atom_len+bin_idx+7] = 0.0;
     
     bin_count[c] += 1; 

  }

  int chk = 0;
  printf("bin_count[%d] = %d\n", chk, bin_count[chk]);
  
  for(i = 0; i < bin_count[chk]; i++){
     //printf("c = %d, ipart = %f, Z = %f, x = %f, y = %f, z = %f\n",chk, bin_atom[chk][i*8], bin_atom[chk][i*8+1], bin_atom[chk][i*8+2], bin_atom[chk][i*8+3], bin_atom[chk][i*8+4], bin_atom[chk][i*8+5], bin_atom[chk][i*8+6], bin_atom[chk][i*8+7]);
  }
  printf("-------\n");
  for(i = 0; i < bin_count[chk]; i++){
     //printf("c = %d, ipart = %f, Z = %f, x = %f, y = %f, z = %f\n",chk, bin_atom_ln[chk*bin_atom_len + i*8], bin_atom_ln[chk*bin_atom_len + i*8+1], bin_atom_ln[chk*bin_atom_len + i*8+2], bin_atom_ln[chk*bin_atom_len+i*8+3], bin_atom_ln[chk*bin_atom_len+i*8+4], bin_atom_ln[chk*bin_atom_len+i*8+5], bin_atom_ln[chk*bin_atom_len+i*8+6], bin_atom_ln[chk*bin_atom_len+i*8+7]);
  }


  int cN, cp, cxN, cyN, czN, cxsh, cysh, czsh;
  float rsh_x, rsh_y, rsh_z;
  float nl_list[Ncell][27*4];
  //float nl_list_ln[Ncell*27*4];
  float *nl_list_ln = (float *)malloc(Ncell*27*4*sizeof(float));
  const int cell_len = 27*4;
  int n_ct;

  for(c = 0; c < Ncell; c++){
     cz = c/(Lxd*Lyd);
     cp = c % (Lxd*Lyd);
     cy = cp/Lxd;
     cx = cp % Lxd;

     n_ct = 0;
     for(czN = cz-1; czN < cz+2; czN++){
       
        if(czN < 0){
          czsh = Lzd;
          rsh_z = -Lz;
        }
        else if(czN >= Lzd){
          czsh = -Lzd;
          rsh_z = Lz;
        }
        else{
          czsh = 0;
          rsh_z = 0;
        }

        for(cyN = cy-1; cyN < cy+2; cyN++){
       
           if(cyN < 0){
             cysh = Lyd;
             rsh_y = -Ly;
           }
           else if(cyN >= Lyd){
             cysh = -Lyd;
             rsh_y = Ly;
           }
           else{
             cysh = 0;
             rsh_y = 0;
           }


           for(cxN = cx-1; cxN < cx+2; cxN++){
       
              if(cxN < 0){
                cxsh = Lxd;
                rsh_x = -Lx;
              }
              else if(cxN >= Lxd){
                cxsh = -Lxd;
                rsh_x = Lx;
              }
              else{
                cxsh = 0;
                rsh_x = 0;
              }

              cN = cxN + cxsh + (cyN + cysh)*Lxd + (czN + czsh)*Lxd*Lyd;
              nl_list[c][n_ct] = cN;
              nl_list[c][n_ct+1] = rsh_x;
              nl_list[c][n_ct+2] = rsh_y;
              nl_list[c][n_ct+3] = rsh_z;
             
              nl_list_ln[c*cell_len+n_ct] = cN;
              nl_list_ln[c*cell_len+n_ct+1] = rsh_x;
              nl_list_ln[c*cell_len+n_ct+2] = rsh_y;
              nl_list_ln[c*cell_len+n_ct+3] = rsh_z;
 
              n_ct += 4;


            }

          }

       } 

  }





  int nchk = 56;
  for(int inl = 0; inl < 27; inl++){
     //printf("c = %d, cN = %f, rxsh = %f, rysh = %f, rzsh = %f\n", nchk, nl_list[nchk][inl*4], nl_list[nchk][inl*4+1], nl_list[nchk][inl*4+2], nl_list[nchk][inl*4+3]);
     //printf("c = %d, cN = %f, rxsh = %f, rysh = %f, rzsh = %f\n", nchk, nl_list_ln[nchk*cell_len+inl*4], nl_list_ln[nchk*cell_len+inl*4+1], nl_list_ln[nchk*cell_len+inl*4+2], nl_list_ln[nchk*cell_len+inl*4+3]);
  }
  printf("-----------\n");
  for(int inl = 0; inl < 27; inl++){
     //printf("c = %d, cN = %f, rxsh = %f, rysh = %f, rzsh = %f\n", nchk, nl_list[nchk][inl*4], nl_list[nchk][inl*4+1], nl_list[nchk][inl*4+2], nl_list[nchk][inl*4+3]);
     //printf("c = %d, cN = %f, rxsh = %f, rysh = %f, rzsh = %f\n", nchk, nl_list_ln[nchk*cell_len+inl*4], nl_list_ln[nchk*cell_len+inl*4+1], nl_list_ln[nchk*cell_len+inl*4+2], nl_list_ln[nchk*cell_len+inl*4+3]);
  }

  clock_gettime(CLOCK_MONOTONIC, &start_c);

  int jn;
  float Zc_ip, x_ip, y_ip, z_ip; 
  float Zc_jp, x_jp, y_jp, z_jp;
  int c_ipart, c_jpart; 
  float n_x_sh, n_y_sh, n_z_sh;
  float dx, dy, dz, r, f1, f2, f3, fr;
  int ip_cn = 0;

  for(int ic = 0; ic < Ncell; ic++){
     for(int ip = 0; ip < bin_count[ic]; ip++){
       
        c_ipart = bin_atom[ic][ip*8];
        Zc_ip = bin_atom[ic][ip*8+1];
        x_ip = bin_atom[ic][ip*8+2];
        y_ip = bin_atom[ic][ip*8+3];
        z_ip = bin_atom[ic][ip*8+4];

        for(int jc = 0; jc < 27; jc++){
           jn = (int) nl_list[ic][jc*4];
           for(int jp = 0; jp < bin_count[jn]; jp++){

              c_jpart = bin_atom[jn][jp*8];
              Zc_jp = bin_atom[jn][jp*8+1];
              x_jp = bin_atom[jn][jp*8+2];
              y_jp = bin_atom[jn][jp*8+3];
              z_jp = bin_atom[jn][jp*8+4];
              
              n_x_sh = nl_list[ic][jc*4+1];         
              n_y_sh = nl_list[ic][jc*4+2];         
              n_z_sh = nl_list[ic][jc*4+3];         
              //if(ic == 0) {printf("c = %d, c_i = %d, c_j = %d\n", ic, c_ipart, c_jpart);}
              if(c_ipart != c_jpart){
              
                //if((ic == 0) && (ip == 0)) {printf("c = %d, c_i = %d, c_j = %d\n", ic, c_ipart, c_jpart); ip_cn += 1;}
                dx = x_ip - (x_jp + n_x_sh);
                dy = y_ip - (y_jp + n_y_sh);
                dz = z_ip - (z_jp + n_z_sh);
                r = sqrt(dx*dx + dy*dy + dz*dz);

                if(r < rc){
                  f1 = (0.5/(r*r)) * exp(kappa*r) * erfc(G*r + 0.5*kappa/G) * (1.0 - kappa*r);
                  f2 = (0.5/(r*r)) * exp(-kappa*r) * erfc(G*r - 0.5*kappa/G) * (1.0 + kappa*r);
                  f3 = (G/(sqrt(pi)*r)) * (exp(-pow((G*r + 0.5*kappa/G),2)) * exp(kappa*r) + exp(-pow((G*r - 0.5*kappa/G),2)) * exp(-kappa*r) );
                  fr = Zc_ip*Zc_jp*(f1 + f2 + f3);           
               
                  bin_atom[ic][ip*8+5] = bin_atom[ic][ip*8+5] + (fr*dx/r);   
                  bin_atom[ic][ip*8+6] = bin_atom[ic][ip*8+6] + (fr*dy/r);   
                  bin_atom[ic][ip*8+7] = bin_atom[ic][ip*8+7] + (fr*dz/r);   
                  //bin_atom[jn][jp*8+5] = bin_atom[jn][jp*8+5] - (fr*dx/r);   
                  //bin_atom[jn][jp*8+6] = bin_atom[jn][jp*8+6] - (fr*dy/r);   
                  //bin_atom[jn][jp*8+7] = bin_atom[jn][jp*8+7] - (fr*dz/r);   

                }

              }    


           }


        }

     }

  }
  clock_gettime(CLOCK_MONOTONIC, &end_c);
  diff_c = (end_c.tv_sec - start_c.tv_sec)*1000000.0 + (end_c.tv_nsec - start_c.tv_nsec)/1000.0;
  printf("elapsed time = %lf micro-seconds\n", diff_c);

  printf("no. of particles for c = 0, ip = 0: %d\n", ip_cn);

  //update(kappa, G, rc, pi, Ncell, bin_count, bin_atom_ln, bin_atom_len, nl_list_ln, cell_len);

  for(int cchk = 0; cchk < Ncell; cchk++){ 
     for(i = 0; i < bin_count[cchk]; i++){
     if(bin_atom[cchk][i*8] < 10) 
       {printf("c = %d, ipart = %f, Z = %f, x = %f, y = %f, z = %f, ax = %f, ay = %f, az = %f\n",cchk, bin_atom[cchk][i*8], bin_atom[cchk][i*8+1], bin_atom[cchk][i*8+2], bin_atom[cchk][i*8+3], bin_atom[cchk][i*8+4], bin_atom[cchk][i*8+5], bin_atom[cchk][i*8+6], bin_atom[cchk][i*8+7]);}
     }
  }
  printf("-------------------\n");
  for(int cchk = 0; cchk < Ncell; cchk++){ 
     for(i = 0; i < bin_count[cchk]; i++){
     if(bin_atom[cchk][i*8] > (N-10)) 
       {printf("c = %d, ipart = %f, Z = %f, x = %f, y = %f, z = %f, ax = %f, ay = %f, az = %f\n",cchk, bin_atom[cchk][i*8], bin_atom[cchk][i*8+1], bin_atom[cchk][i*8+2], bin_atom[cchk][i*8+3], bin_atom[cchk][i*8+4], bin_atom[cchk][i*8+5], bin_atom[cchk][i*8+6], bin_atom[cchk][i*8+7]);}
     }
  }

  printf("-------------\n");

  float *d_bin_atom_ln, *d_nl_list_ln;
  int *d_bin_count;
  clock_gettime(CLOCK_MONOTONIC, &start);
  
  hipMalloc((void **) &d_bin_count, Ncell*sizeof(int));
  hipMalloc((void **) &d_bin_atom_ln, Ncell*bin_atom_len*sizeof(float));
  hipMalloc((void **) &d_nl_list_ln, Ncell*27*4*sizeof(float));

  dim3 dimGrid(512, ceil(Ncell/512.0), 1);
  dim3 dimBlock(64, 1, 1);
  //clock_gettime(CLOCK_MONOTONIC, &start);
  
  hipMemcpy(d_bin_count, bin_count, Ncell*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_bin_atom_ln, bin_atom_ln, Ncell*bin_atom_len*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_nl_list_ln, nl_list_ln, Ncell*27*4*sizeof(float), hipMemcpyHostToDevice);

  //clock_gettime(CLOCK_MONOTONIC, &start);
  updateKernel<<<dimGrid, dimBlock>>>(kappa, G, rc, pi, Ncell, d_bin_count, d_bin_atom_ln, bin_atom_len, d_nl_list_ln, cell_len);
  //updateKernel<<<ceil(Ncell/64), 64>>>(kappa, G, rc, pi, Ncell, d_bin_count, d_bin_atom_ln, bin_atom_len, d_nl_list_ln, cell_len);
  //updateKernel<<<ceil(Ncell/128.0), 128>>>();
  hipDeviceSynchronize(); 
  //clock_gettime(CLOCK_MONOTONIC, &end);
  //diff = (end.tv_sec - start.tv_sec)*1000000.0 + (end.tv_nsec - start.tv_nsec)/1000.0;
  //printf("elapsed time = %lf micro-seconds\n", diff);

  hipMemcpy(bin_atom_ln, d_bin_atom_ln, Ncell*bin_atom_len*sizeof(float), hipMemcpyDeviceToHost);

  clock_gettime(CLOCK_MONOTONIC, &end);
  diff = (end.tv_sec - start.tv_sec)*1000000.0 + (end.tv_nsec - start.tv_nsec)/1000.0;
  printf("elapsed time = %lf micro-seconds\n", diff);

  for(int cchk = 0; cchk < Ncell; cchk++){ 
     for(i = 0; i < bin_count[cchk]; i++){
     if(bin_atom_ln[cchk*bin_atom_len+i*8] < 10) 
       {printf("c = %d, ipart = %f, Z = %f, x = %f, y = %f, z = %f, ax = %f, ay = %f, az = %f\n",cchk, bin_atom_ln[cchk*bin_atom_len+i*8], bin_atom_ln[cchk*bin_atom_len+i*8+1], bin_atom_ln[cchk*bin_atom_len+i*8+2], bin_atom_ln[cchk*bin_atom_len+i*8+3], bin_atom_ln[cchk*bin_atom_len+i*8+4], bin_atom_ln[cchk*bin_atom_len+i*8+5], bin_atom_ln[cchk*bin_atom_len+i*8+6], bin_atom_ln[cchk*bin_atom_len+i*8+7]);}
     }
  }
  
  printf("----------------\n");
 
  for(int cchk = 0; cchk < Ncell; cchk++){ 
     for(i = 0; i < bin_count[cchk]; i++){
     if(bin_atom_ln[cchk*bin_atom_len+i*8] > (N-10)) 
       {printf("c = %d, ipart = %f, Z = %f, x = %f, y = %f, z = %f, ax = %f, ay = %f, az = %f\n",cchk, bin_atom_ln[cchk*bin_atom_len+i*8], bin_atom_ln[cchk*bin_atom_len+i*8+1], bin_atom_ln[cchk*bin_atom_len+i*8+2], bin_atom_ln[cchk*bin_atom_len+i*8+3], bin_atom_ln[cchk*bin_atom_len+i*8+4], bin_atom_ln[cchk*bin_atom_len+i*8+5], bin_atom_ln[cchk*bin_atom_len+i*8+6], bin_atom_ln[cchk*bin_atom_len+i*8+7]);}
     }
  }

  hipFree(d_bin_atom_ln); hipFree(d_nl_list_ln); hipFree(d_bin_count);

  free(pos); free(Z);
  free(bin_count); free(bin_atom_ln); free(nl_list_ln);
  
  return 0;
} 
